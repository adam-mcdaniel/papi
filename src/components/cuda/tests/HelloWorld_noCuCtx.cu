#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/**
 * @file    HelloWorld_noCuCtx.cu
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:	Anustuv Pal
 *			anustuv@icl.utk.edu
 * Mods:	<your name here>
 *			<your email address>
 * test case for cuda component
 *
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *	with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *	of the component tests (if all tests are added to a directory named
 *	'tests' in the specific component dir).
 *	See components/README for more details.
 *
 *	The string "Hello World!" is mangled and then restored.
 *
 *  CUDA Context notes for CUPTI_11: Although a hipSetDevice() will create a
 *  primary context for the device that allows kernel execution; PAPI cannot
 *  use a primary context to control the Nvidia Performance Profiler.
 *  Applications must create a context using hipCtxCreate() that will execute
 *  the kernel, this must be done prior to the PAPI_add_events() invocation in
 *  the code below. If multiple GPUs are in use, each requires its own context,
 *  and that context should be active when PAPI_events are added for each
 *  device.  Which means using Seperate PAPI_add_events() for each device. For
 *  an example see simpleMultiGPU.cu.
 *
 *  There are three points below where hipCtxCreate() is called, this code works
 *  if any one of them is used alone.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#ifdef PAPI
#include "papi.h"
#include "papi_test.h"
#endif

#define STEP_BY_STEP_DEBUG 0 /* helps debug hipCtx_t issues. */
#define PRINT(quiet, format, args...) {if (!quiet) {fprintf(stderr, format, ## args);}}

// Device kernel
__global__ void
helloWorld(char* str)
{
        // determine where in the thread grid we are
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        // unmangle output
        str[idx] += idx;
}

/** @class add_events_from_command_line
  * @brief Try and add each event provided on the command line by the user.
  *
  * @param EventSet
  *   A PAPI eventset.
  * @param totalEventCount
  *   Number of events from the command line.
  * @param eventNamesFromCommandLine
  *   Events provided on the command line.
  * @param *numEventsSuccessfullyAdded
  *   Total number of successfully added events.
  * @param **eventsSuccessfullyAdded
  *   Events that we are able to add to the EventSet.
  * @param *numMultipassEvents
  *   Counter to see if a multiple pass event was provided on the command line.
*/
static void add_events_from_command_line(int EventSet, int totalEventCount, char **eventNamesFromCommandLine, int *numEventsSuccessfullyAdded, char **eventsSuccessfullyAdded, int *numMultipassEvents)
{
    int i;
    for (i = 0; i < totalEventCount; i++) {
        int papi_errno = PAPI_add_named_event(EventSet, eventNamesFromCommandLine[i]);
        if (papi_errno != PAPI_OK) {
            if (papi_errno != PAPI_EMULPASS) {
                fprintf(stderr, "Unable to add event %s to the EventSet with error code %d.\n", eventNamesFromCommandLine[i], papi_errno);
                test_skip(__FILE__, __LINE__, "", 0);
            }

            // Handle multiple pass events
            (*numMultipassEvents)++;
            continue;
        }

        // Handle successfully added events
        int strLen = snprintf(eventsSuccessfullyAdded[(*numEventsSuccessfullyAdded)], PAPI_MAX_STR_LEN, "%s", eventNamesFromCommandLine[i]);
        if (strLen < 0 || strLen >= PAPI_MAX_STR_LEN) {
            fprintf(stderr, "Failed to fully write successfully added event.\n");
            test_skip(__FILE__, __LINE__, "", 0);
        }
        (*numEventsSuccessfullyAdded)++;
    }

    return;
}

// Host function
int main(int argc, char** argv)
{
    int quiet = 0;
    hipError_t hipError_t;
    hipError_t cuError; (void) cuError;

    hipInit(0);

#ifdef PAPI
    char *test_quiet = getenv("PAPI_CUDA_TEST_QUIET");
    if (test_quiet)
        quiet = (int) strtol(test_quiet, (char**) NULL, 10);

    /* PAPI Initialization */
    int papi_errno = PAPI_library_init( PAPI_VER_CURRENT );
    if( papi_errno != PAPI_VER_CURRENT ) {
        test_fail(__FILE__,__LINE__, "PAPI_library_init failed", 0);
    }

    printf( "PAPI_VERSION     : %4d %6d %7d\n",
        PAPI_VERSION_MAJOR( PAPI_VERSION ),
        PAPI_VERSION_MINOR( PAPI_VERSION ),
        PAPI_VERSION_REVISION( PAPI_VERSION ) );

    int i;
    int EventSet = PAPI_NULL;
    int eventCount = argc - 1;

    /* if no events passed at command line, just report test skipped. */
    if (eventCount == 0) {
        fprintf(stderr, "No events specified at command line.");
        test_skip(__FILE__,__LINE__, "", 0);
    }

    long long *values = (long long *) calloc(eventCount, sizeof (long long));
    if (values == NULL) {
       test_fail(__FILE__, __LINE__, "Failed to allocate memory for values.\n", 0);
    }

    int *events = (int *) calloc(eventCount, sizeof (int));
    if (events == NULL) {
        test_fail(__FILE__, __LINE__, "Failed to allocate memory for events.\n", 0);
    }

    papi_errno = PAPI_create_eventset( &EventSet );
    if( papi_errno != PAPI_OK ) {
        test_fail(__FILE__,__LINE__,"Cannot create eventset",papi_errno);
    }

    // Handle the events from the command line
    int numEventsSuccessfullyAdded = 0, numMultipassEvents = 0;
    char **eventsSuccessfullyAdded, **metricNames = argv + 1;
    eventsSuccessfullyAdded = (char **) malloc(eventCount * sizeof(char *));
    if (eventsSuccessfullyAdded == NULL) {
        fprintf(stderr, "Failed to allocate memory for successfully added events.\n");
        test_skip(__FILE__, __LINE__, "", 0);
    }
    for (i = 0; i < eventCount; i++) {
        eventsSuccessfullyAdded[i] = (char *) malloc(PAPI_MAX_STR_LEN * sizeof(char));
        if (eventsSuccessfullyAdded[i] == NULL) {
            fprintf(stderr, "Failed to allocate memory for command line argument.\n");
            test_skip(__FILE__, __LINE__, "", 0);
        }
    }

    add_events_from_command_line(EventSet, eventCount, metricNames, &numEventsSuccessfullyAdded, eventsSuccessfullyAdded, &numMultipassEvents);

    // Only multiple pass events were provided on the command line
    if (numEventsSuccessfullyAdded == 0) {
        fprintf(stderr, "Events provided on the command line could not be added to an EventSet as they require multiple passes.\n");
        test_skip(__FILE__, __LINE__, "", 0);
    }

    papi_errno = PAPI_start( EventSet );
    if( papi_errno != PAPI_OK ) {
        test_fail(__FILE__, __LINE__, "PAPI_start failed.", papi_errno);
    }

#endif

    int j;

    // desired output
    char str[] = "Hello World!";

    // mangle contents of output
    // the null character is left intact for simplicity
    for(j = 0; j < 12; j++) {
        str[j] -= j;
    }

    PRINT( quiet, "mangled str=%s\n", str );

    // allocate memory on the device
    char *d_str;
    size_t size = sizeof(str);
    hipMalloc((void**)&d_str, size);

    // copy the string to the device
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

    // set the grid and block sizes
    dim3 dimGrid(2); // one block per word
    dim3 dimBlock(6); // one thread per character

    // invoke the kernel
    helloWorld<<< dimGrid, dimBlock >>>(d_str);

    hipError_t = hipGetLastError();
    if (STEP_BY_STEP_DEBUG) {
        fprintf(stderr, "%s:%s:%i Kernel Return Code: %s.\n", __FILE__, __func__, __LINE__, hipGetErrorString(hipError_t));
    }

    // retrieve the results from the device
    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

    // free up the allocated memory on the device
    hipFree(d_str);

#ifdef PAPI
    papi_errno = PAPI_read( EventSet, values );
    if( papi_errno != PAPI_OK ) {
        test_fail(__FILE__, __LINE__, "PAPI_read failed", papi_errno);
    }

    for( i = 0; i < numEventsSuccessfullyAdded; i++ ) {
        PRINT( quiet, "read: %12lld \t=0X%016llX \t\t --> %s \n", values[i], values[i], eventsSuccessfullyAdded[i] );
    }

    papi_errno = PAPI_stop( EventSet, values );
    if( papi_errno != PAPI_OK ) {
        test_fail(__FILE__, __LINE__, "PAPI_stop failed", papi_errno);
    }

    papi_errno = PAPI_cleanup_eventset(EventSet);
    if( papi_errno != PAPI_OK ) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed", papi_errno);
    }

    papi_errno = PAPI_destroy_eventset(&EventSet);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed", papi_errno);
    }

    for( i = 0; i < numEventsSuccessfullyAdded; i++ ) {
        PRINT( quiet, "stop: %12lld \t=0X%016llX \t\t --> %s \n", values[i], values[i], eventsSuccessfullyAdded[i] );
    }

    // Free allocated memory
    free(values);
    free(events); 
    for (i = 0; i < eventCount; i++) {
        free(eventsSuccessfullyAdded[i]);
    }
    free(eventsSuccessfullyAdded);

    PAPI_shutdown();

    // Output a note that a multiple pass event was provided on the command line
    if (numMultipassEvents > 0) {
        PRINT(quiet, "\033[0;33mNOTE: From the events provided on the command line, an event or events requiring multiple passes was detected and not added to the EventSet. Check your events with utils/papi_native_avail.\n\033[0m");
    }

    test_pass(__FILE__);
#endif

	return 0;
}
